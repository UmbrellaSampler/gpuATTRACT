#include "as/DeviceDataInterface.h"
#include "asUtils/macros.h"
#include "config.h"



__constant__ as::deviceGridUnionDesc c_Grids[DEVICE_MAXGRIDS];
__constant__ as::deviceProteinDesc c_Proteins[DEVICE_MAXPROTEINS];
__constant__ as::deviceParamTableDesc c_ParamTable;
__constant__ as::deviceSimParam c_SimParam;

void as::setDeviceGridUnion(const deviceGridUnionDesc &desc,
		unsigned deviceId, unsigned localDeviceID)
{
	CUDA_CHECK(hipSetDevice(deviceId));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_Grids), &desc,
			sizeof(deviceGridUnionDesc),
			localDeviceID*sizeof(deviceGridUnionDesc), hipMemcpyHostToDevice));
}

void as::unsetDeviceGridUnion(
		unsigned deviceId, unsigned localDeviceID)
{
	CUDA_CHECK(hipSetDevice(deviceId));
	deviceGridUnionDesc desc;
	memset(&desc, 0, sizeof(deviceGridUnionDesc));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_Grids), &desc,
			sizeof(deviceGridUnionDesc),
			localDeviceID*sizeof(deviceGridUnionDesc), hipMemcpyHostToDevice));
}

void as::setDeviceProtein(const deviceProteinDesc &desc,
		unsigned deviceId, unsigned localDeviceID)
{
	CUDA_CHECK(hipSetDevice(deviceId));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_Proteins), &desc,
			sizeof(deviceProteinDesc),
			localDeviceID*sizeof(deviceProteinDesc), hipMemcpyHostToDevice));
}

void as::unsetDeviceProtein(
		unsigned deviceId, unsigned localDeviceID)
{
	CUDA_CHECK(hipSetDevice(deviceId));
	deviceProteinDesc desc;
	memset(&desc, 0, sizeof(deviceProteinDesc));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_Proteins), &desc,
			sizeof(deviceProteinDesc),
			localDeviceID*sizeof(deviceProteinDesc), hipMemcpyHostToDevice));
}

void as::setDeviceParamTable(const deviceParamTableDesc& desc,
		unsigned deviceId)
{
	CUDA_CHECK(hipSetDevice(deviceId));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_ParamTable), &desc,
			sizeof(deviceParamTableDesc), 0, hipMemcpyHostToDevice));
}

void as::unsetDeviceParamTable(unsigned deviceId)
{
	CUDA_CHECK(hipSetDevice(deviceId));
	deviceParamTableDesc desc;
	memset(&desc, 0, sizeof(deviceParamTableDesc));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_ParamTable), &desc,
			sizeof(deviceParamTableDesc), 0, hipMemcpyHostToDevice));
}

void as::setDeviceSimParam(const SimParam& simPar,
		unsigned deviceId)
{
	CUDA_CHECK(hipSetDevice(deviceId));
	deviceSimParam desc;
	desc.dielec 	= simPar.dielec;
	desc.epsilon	= simPar.epsilon;
	desc.ffelec		= simPar.ffelec;
	desc.useRecGrad	= simPar.useRecGrad;
	desc.usePot		= simPar.usePot;
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_SimParam), &desc,
			sizeof(c_SimParam), 0, hipMemcpyHostToDevice));
}

void as::unsetDeviceSimParam(unsigned deviceId)
{
	CUDA_CHECK(hipSetDevice(deviceId));
	deviceSimParam desc;
	memset(&desc, 0, sizeof(deviceSimParam));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_SimParam), &desc,
			sizeof(c_SimParam), 0, hipMemcpyHostToDevice));
}

