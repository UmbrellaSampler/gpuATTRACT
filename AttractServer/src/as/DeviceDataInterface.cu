#include "as/DeviceDataInterface.h"
#include "asUtils/macros.h"
#include "config.h"



__constant__ as::deviceGridUnionDesc c_Grids[DEVICE_MAXGRIDS];
__constant__ as::deviceProteinDesc c_Proteins[DEVICE_MAXPROTEINS];
__constant__ as::deviceParamTableDesc c_ParamTable;
__constant__ as::deviceSimParam c_SimParam;

void as::setDeviceGridUnion(const deviceGridUnionDesc &desc,
		unsigned deviceId, unsigned localDeviceID)
{
	cudaVerify(hipSetDevice(deviceId));
	cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(c_Grids), &desc,
			sizeof(deviceGridUnionDesc),
			localDeviceID*sizeof(deviceGridUnionDesc), hipMemcpyHostToDevice));
}

void as::unsetDeviceGridUnion(
		unsigned deviceId, unsigned localDeviceID)
{
	cudaVerify(hipSetDevice(deviceId));
	deviceGridUnionDesc desc;
	memset(&desc, 0, sizeof(deviceGridUnionDesc));
	cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(c_Grids), &desc,
			sizeof(deviceGridUnionDesc),
			localDeviceID*sizeof(deviceGridUnionDesc), hipMemcpyHostToDevice));
}

void as::setDeviceProtein(const deviceProteinDesc &desc,
		unsigned deviceId, unsigned localDeviceID)
{
	cudaVerify(hipSetDevice(deviceId));
	cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(c_Proteins), &desc,
			sizeof(deviceProteinDesc),
			localDeviceID*sizeof(deviceProteinDesc), hipMemcpyHostToDevice));
}

void as::unsetDeviceProtein(
		unsigned deviceId, unsigned localDeviceID)
{
	cudaVerify(hipSetDevice(deviceId));
	deviceProteinDesc desc;
	memset(&desc, 0, sizeof(deviceProteinDesc));
	cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(c_Proteins), &desc,
			sizeof(deviceProteinDesc),
			localDeviceID*sizeof(deviceProteinDesc), hipMemcpyHostToDevice));
}

void as::setDeviceParamTable(const deviceParamTableDesc& desc,
		unsigned deviceId)
{
	cudaVerify(hipSetDevice(deviceId));
	cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(c_ParamTable), &desc,
			sizeof(deviceParamTableDesc), 0, hipMemcpyHostToDevice));
}

void as::unsetDeviceParamTable(unsigned deviceId)
{
	cudaVerify(hipSetDevice(deviceId));
	deviceParamTableDesc desc;
	memset(&desc, 0, sizeof(deviceParamTableDesc));
	cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(c_ParamTable), &desc,
			sizeof(deviceParamTableDesc), 0, hipMemcpyHostToDevice));
}

void as::setDeviceSimParam(const SimParam& simPar,
		unsigned deviceId)
{
	cudaVerify(hipSetDevice(deviceId));
	deviceSimParam desc;
	desc.dielec 	= simPar.dielec;
	desc.epsilon	= simPar.epsilon;
	desc.ffelec		= simPar.ffelec;
	desc.useRecGrad	= simPar.useRecGrad;
	desc.usePot		= simPar.usePot;
	cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(c_SimParam), &desc,
			sizeof(c_SimParam), 0, hipMemcpyHostToDevice));
}

void as::unsetDeviceSimParam(unsigned deviceId)
{
	cudaVerify(hipSetDevice(deviceId));
	deviceSimParam desc;
	memset(&desc, 0, sizeof(deviceSimParam));
	cudaVerify(hipMemcpyToSymbol(HIP_SYMBOL(c_SimParam), &desc,
			sizeof(c_SimParam), 0, hipMemcpyHostToDevice));
}

